#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <iostream>
#include <random>
#include <fstream>

// set a 3D volume
// To compile it with nvcc execute: nvcc -O2 -o set3d set3d.cu
//define the data set size (cubic volume)
#define DATAXSIZE 128
#define DATAYSIZE 128
#define DATAZSIZE 128
//define the chunk sizes that each threadblock will work on
#define BLKXSIZE 32
#define BLKYSIZE 4
#define BLKZSIZE 4

using namespace std;

// for cuda error checking
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            return 1; \
        } \
    } while (0)

__device__ double Laplacian(double c[][DATAYSIZE][DATAXSIZE], double dx, double dy, double dz, int x, int y, int z)
{

  int xp, xn, yp, yn, zp, zn;

  int nx = (int)DATAXSIZE - 1;
  int ny = (int)DATAYSIZE - 1;
  int nz = (int)DATAZSIZE - 1;

  xp = x+1;
  xn = x-1;
  yp = y+1;
  yn = y-1;
  zp = z+1;
  zn = z-1;

  if (xp > nx)
  {
   xp = 0;
  }
  if (xn < 0)
  {
   xn = nx;
  }
  
  if (yp > ny)
  {
   yp = 0;
  }
  if (yn < 0)
  {
   yn = ny;
  }

  if (zp > nz)
  {
   zp = 0;
  }
  if (zn < 0)
  {
   zn = nz;
  }

  double cxx = (c[xp][y][z] + c[xn][y][z] - 2.0*c[x][y][z]) / (dx*dx);
  double cyy = (c[x][yp][z] + c[x][yn][z] - 2.0*c[x][y][z]) / (dy*dy);
  double czz = (c[x][y][zp] + c[x][y][zn] - 2.0*c[x][y][z]) / (dz*dz);

  double result = cxx + cyy + czz;

  return result;

}

__device__ double GradientX(double phi[][DATAYSIZE][DATAXSIZE], double dx, double dy, double dz, int x, int y, int z)
{

  int xp, xn;

  int nx = (int)DATAXSIZE - 1;

  xp = x+1;
  xn = x-1;

  if (xp > nx)
  {
   xp = 0;
  }
  if (xn < 0)
  {
   xn = nx;
  }

  double phix = (phi[xp][y][z] - phi[xn][y][z]) / (2.0*dx);

  return phix;

}

__device__ double GradientY(double phi[][DATAYSIZE][DATAXSIZE], double dx, double dy, double dz, int x, int y, int z)
{

  int yp, yn;

  int ny = (int)DATAYSIZE - 1;

  yp = y+1;
  yn = y-1;

  if (yp > ny)
  {
   yp = 0;
  }
  if (yn < 0)
  {
   yn = ny;
  }

  double phiy = (phi[x][yp][z] - phi[x][yn][z]) / (2.0*dy);

  return phiy;

}

__device__ double GradientZ(double phi[][DATAYSIZE][DATAXSIZE], double dx, double dy, double dz, int x, int y, int z)
{

  int zp, zn;

  int nz = (int)DATAZSIZE - 1;

  zp = z+1;
  zn = z-1;

  if (zp > nz)
  {
   zp = 0;
  }
  if (zn < 0)
  {
   zn = nz;
  }

  double phiz = (phi[x][y][zp] - phi[x][y][zn]) / (2.0*dz);

  return phiz;

}

__global__ void chemicalPotential(double c[][DATAYSIZE][DATAXSIZE], double mu[][DATAYSIZE][DATAXSIZE], double dx, double dy, double dz, double gamma, double e_AA, double e_BB, double e_AB)
{

 unsigned idx = blockIdx.x*blockDim.x + threadIdx.x;
 unsigned idy = blockIdx.y*blockDim.y + threadIdx.y;
 unsigned idz = blockIdx.z*blockDim.z + threadIdx.z;

 if ((idx < (DATAXSIZE)) && (idy < (DATAYSIZE)) && (idz < (DATAZSIZE))){

  mu[idx][idy][idz] = ( 9.0 / 2.0 )*( ( c[idx][idy][idz] + 1.0 ) * e_AA + ( c[idx][idy][idz] - 1 ) * e_BB - 2.0 * c[idx][idy][idz] * e_AB ) + 3.0 * c[idx][idy][idz] + c[idx][idy][idz] * c[idx][idy][idz] * c[idx][idy][idz] - gamma * Laplacian(c,dx,dy,dz,idx,idy,idz);
 }

}

__device__ double freeEnergy(double c, double e_AA, double e_BB, double e_AB)
{

 return (((9.0 / 4.0) * ((c*c+2.0*c+1.0)*e_AA+(c*c-2.0*c+1.0)*e_BB+2.0*(1.0-c*c)*e_AB)) + ((3.0/2.0) * c * c) + ((3.0/12.0) * c * c * c * c));

}

__global__ void localFreeEnergyFunctional(double c[][DATAYSIZE][DATAXSIZE], double f[][DATAYSIZE][DATAXSIZE], double dx, double dy, double dz, double gamma, double e_AA, double e_BB, double e_AB)
{

 unsigned idx = blockIdx.x*blockDim.x + threadIdx.x;
 unsigned idy = blockIdx.y*blockDim.y + threadIdx.y;
 unsigned idz = blockIdx.z*blockDim.z + threadIdx.z;

 if ((idx < (DATAXSIZE)) && (idy < (DATAYSIZE)) && (idz < (DATAZSIZE))){

  f[idx][idy][idz] = freeEnergy(c[idx][idy][idz],e_AA,e_BB,e_AB) + (gamma / 2.0) * (GradientX(c,dx,dy,dz,idx,idy,idz) * GradientX(c,dx,dy,dz,idx,idy,idz) + GradientY(c,dx,dy,dz,idx,idy,idz) * GradientY(c,dx,dy,dz,idx,idy,idz) + GradientZ(c,dx,dy,dz,idx,idy,idz) * GradientZ(c,dx,dy,dz,idx,idy,idz));
 }

}

// device function to set the 3D volume
__global__ void cahnHilliard(double cnew[][DATAYSIZE][DATAXSIZE], double cold[][DATAYSIZE][DATAXSIZE], double mu[][DATAYSIZE][DATAXSIZE], double D, double dt, double dx, double dy, double dz)
{
    unsigned idx = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned idy = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned idz = blockIdx.z*blockDim.z + threadIdx.z;
    if ((idx < (DATAXSIZE)) && (idy < (DATAYSIZE)) && (idz < (DATAZSIZE))){
      cnew[idx][idy][idz] = cold[idx][idy][idz] + dt * D * Laplacian(mu,dx,dy,dz,idx,idy,idz);
      }
}

__global__ void Swap(double cnew[][DATAYSIZE][DATAXSIZE], double cold[][DATAYSIZE][DATAXSIZE])
{
    unsigned idx = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned idy = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned idz = blockIdx.z*blockDim.z + threadIdx.z;
    double tmp;    

    if ((idx < (DATAXSIZE)) && (idy < (DATAYSIZE)) && (idz < (DATAZSIZE))) {
     tmp=cnew[idx][idy][idz];
     cnew[idx][idy][idz]=cold[idx][idy][idz];
     cold[idx][idy][idz]=tmp;
    }

}

void initialization(double c[][DATAYSIZE][DATAXSIZE])
{
    for (unsigned int idx = 0.0; idx < DATAXSIZE; idx++) {
     for (unsigned int idy = 0.0; idy < DATAYSIZE; idy++) {
      for (unsigned int idz = 0.0; idz < DATAZSIZE; idz++) {
      double f = (double)rand() / RAND_MAX;
      c[idx][idy][idz] = -1.0 + 2.0*f;
    }
     }
      }
}

void write_output_vtk(double c[][DATAYSIZE][DATAXSIZE], int t, int nx, int ny, int nz, string output, string variableName)
{
    string name = "./sweep_1/" + output + "_" + to_string(t) + ".vtk";
    ofstream ofile (name);

    // vtk preamble
    ofile << "# vtk DataFile Version 2.0" << endl;
    ofile << "OUTPUT by LIBM\n";
    ofile << "ASCII" << endl;

    // write grid
    ofile << "DATASET RECTILINEAR_GRID" << endl;
    ofile << "DIMENSIONS " << nx << " " << ny << " " << nz << endl;
    ofile << "X_COORDINATES " << nx << " float" << endl;
    for(size_t i = 0; i < nx; i++)
        ofile << i << "\t";
    ofile << endl;
    ofile << "Y_COORDINATES " << ny << " float" << endl;
    for(size_t i = 0; i < ny; i++)
        ofile << i << "\t";
    ofile << endl;
    ofile << "Z_COORDINATES " << nz << " float" << endl;
    for(size_t i = 0; i < nz; i++)
        ofile << i << "\t";
    ofile << endl;

    // point data
    ofile << "POINT_DATA " << nx*ny*nz << endl;

    // write rho
    ofile << "SCALARS " << variableName << " double" << endl;
    ofile << "LOOKUP_TABLE default" << endl;
  for (int k = 0; k < nz; k++) 
    for(int j = 0; j < ny; j++)
        for(int i = 0; i < nx; i++)
            ofile << c[i][j][k] << endl;

}

double integral(double c[][DATAYSIZE][DATAXSIZE], int nx, int ny, int nz)
{
  double summation = 0.0;  

  for (int k = 0; k < nz; k++)
    for(int j = 0; j < ny; j++)
        for(int i = 0; i < nx; i++)
            summation = summation + c[i][j][k];

  return summation;
}

int main(int argc, char *argv[])
{
    int iterator = 0;

    double e_AA = -(7.0/9.0); 

    double e_AA_f = (11.0/9.0);
    
    double deltaEAA = (1.0/9.0);

    while (e_AA < e_AA_f + deltaEAA)
    {

    printf("e_AA is: %f\n",e_AA);

    double dx = 1.0;
    double dy = 1.0;
    double dz = 1.0;
    double dt = 0.01;
    double e_BB = e_AA;
    double e_AB = (2.0/9.0);
    int t_f = 25000;
    int t_freq = 10;
    double gamma = 0.5;
    double D = 1.0;
    string output_c = "outputc" + to_string(iterator);
    string output_mu = "outputmu" + to_string(iterator);
    string output_f = "outputf" + to_string(iterator);
    string variableName_c = "c";
    string variableName_mu = "mu";
    string variableName_f = "f";
    hipSetDevice(0.0);
    typedef double nRarray[DATAYSIZE][DATAXSIZE];
    const dim3 blockSize(BLKXSIZE, BLKYSIZE, BLKZSIZE);
    const dim3 gridSize(((DATAXSIZE+BLKXSIZE-1)/BLKXSIZE), ((DATAYSIZE+BLKYSIZE-1)/BLKYSIZE), ((DATAZSIZE+BLKZSIZE-1)/BLKZSIZE));
// overall data set sizes
    const int nx = DATAXSIZE;
    const int ny = DATAYSIZE;
    const int nz = DATAZSIZE;
// pointers for data set storage via malloc
    nRarray *c_host; // storage for result stored on host
    nRarray *mu_host;
    nRarray *f_host;
    nRarray *d_cold;  // storage for result computed on device
    nRarray *d_cnew;
    nRarray *d_muold;
    nRarray *d_fold;
// allocate storage for data set
    if ((c_host = (nRarray *)malloc((nx*ny*nz)*sizeof(double))) == 0) {fprintf(stderr,"malloc1 Fail \n"); return 1;}
    if ((mu_host = (nRarray *)malloc((nx*ny*nz)*sizeof(double))) == 0) {fprintf(stderr,"malloc1 Fail \n"); return 1;}
    if ((f_host = (nRarray *)malloc((nx*ny*nz)*sizeof(double))) == 0) {fprintf(stderr,"malloc1 Fail \n"); return 1;}
// allocate GPU device buffers
    hipMalloc((void **) &d_cold, (nx*ny*nz)*sizeof(double));
    cudaCheckErrors("Failed to allocate device buffer");
    hipMalloc((void **) &d_cnew, (nx*ny*nz)*sizeof(double));
    cudaCheckErrors("Failed to allocate device buffer");
    hipMalloc((void **) &d_muold, (nx*ny*nz)*sizeof(double));
    cudaCheckErrors("Failed to allocate device buffer");
    hipMalloc((void **) &d_fold, (nx*ny*nz)*sizeof(double));
    cudaCheckErrors("Failed to allocate device buffer");
// compute result

    initialization(c_host);

    write_output_vtk(c_host,0,nx,ny,nz,output_c,variableName_c);

    double integral_c = 0.0;
    double integral_mu = 0.0;
    double integral_f = 0.0;

    string name_c = "./sweep_1/integral_c_"+to_string(iterator)+".txt";
    ofstream ofile_c (name_c);

    string name_mu = "./sweep_1/integral_mu_"+to_string(iterator)+".txt";
    ofstream ofile_mu (name_mu);

    string name_f = "./sweep_1/integral_f_"+to_string(iterator)+".txt";
    ofstream ofile_f (name_f);

    integral_c = integral(c_host,nx,ny,nz);

    ofile_c << 0 << "," << integral_c << endl;

    hipMemcpy(d_cold, c_host, ((nx*ny*nz)*sizeof(double)), hipMemcpyHostToDevice);
    cudaCheckErrors("CUDA memcpy failure");

    double clock_d = double(clock()) / CLOCKS_PER_SEC;

    int t = 0;

    while (t < t_f) {

    //printf("Timestep is: %d\n",t);

    chemicalPotential<<<gridSize,blockSize>>>(d_cold,d_muold,dx,dy,dz,gamma,e_AA,e_BB,e_AB);
    cudaCheckErrors("Kernel launch failure");
    localFreeEnergyFunctional<<<gridSize,blockSize>>>(d_cold,d_fold,dx,dy,dz,gamma,e_AA,e_BB,e_AB);
    cudaCheckErrors("Kernel launch failure");
    cahnHilliard<<<gridSize,blockSize>>>(d_cnew,d_cold,d_muold,D,dt,dx,dy,dz);
    cudaCheckErrors("Kernel launch failure");

    if (t == 0) {

     hipMemcpy(mu_host, d_muold, ((nx*ny*nz)*sizeof(double)), hipMemcpyDeviceToHost);   
     cudaCheckErrors("CUDA memcpy failure");

     hipMemcpy(f_host, d_fold, ((nx*ny*nz)*sizeof(double)), hipMemcpyDeviceToHost);
     cudaCheckErrors("CUDA memcpy failure");

     write_output_vtk(mu_host,t,nx,ny,nz,output_mu,variableName_mu);

     write_output_vtk(f_host,t,nx,ny,nz,output_f,variableName_f);

     integral_mu = integral(mu_host,nx,ny,nz);

     ofile_mu << 0 << "," << integral_mu << endl;

     integral_f = integral(f_host,nx,ny,nz);

     ofile_f << 0 << "," << integral_f << endl;

    }

    if (t % t_freq == 0 && t > 0) {

     hipMemcpy(c_host, d_cnew, ((nx*ny*nz)*sizeof(double)), hipMemcpyDeviceToHost);
     cudaCheckErrors("CUDA memcpy failure");

     hipMemcpy(mu_host, d_muold, ((nx*ny*nz)*sizeof(double)), hipMemcpyDeviceToHost);
     cudaCheckErrors("CUDA memcpy failure");

     hipMemcpy(f_host, d_fold, ((nx*ny*nz)*sizeof(double)), hipMemcpyDeviceToHost);
     cudaCheckErrors("CUDA memcpy failure");

     write_output_vtk(c_host,t,nx,ny,nz,output_c,variableName_c);

     write_output_vtk(mu_host,t,nx,ny,nz,output_mu,variableName_mu);

     write_output_vtk(f_host,t,nx,ny,nz,output_f,variableName_f);

     integral_c = integral(c_host,nx,ny,nz);

     ofile_c << t << "," << integral_c << endl;

     integral_mu = integral(mu_host,nx,ny,nz);

     ofile_mu << t << "," << integral_mu << endl;

     integral_f = integral(f_host,nx,ny,nz);

     ofile_f << t << "," << integral_f << endl;

    }
    
    Swap<<<gridSize,blockSize>>>(d_cnew, d_cold);
    cudaCheckErrors("Kernel launch failure");

    t++;

    }

    hipDeviceSynchronize();
    clock_d = double(clock()) / CLOCKS_PER_SEC - clock_d; 
    printf("GPU time = %.3fms\n",clock_d*1e3);

    free(c_host);
    free(mu_host);
    free(f_host);
    hipFree(d_cold);
    cudaCheckErrors("hipFree fail");
    hipFree(d_cnew);
    cudaCheckErrors("hipFree fail");
    hipFree(d_muold);
    cudaCheckErrors("hipFree fail");
    hipFree(d_fold);
    cudaCheckErrors("hipFree fail"); 

    iterator++;
    e_AA += deltaEAA;
    }
    return 0;
}
